#include <hip/hip_runtime.h>

#define TILE_SIZE 32

__global__ void gemm_tiled(const float* A, const float* B, float* C, int N) {
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < N && t * TILE_SIZE + threadIdx.x < N)
            tile_A[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            tile_A[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && t * TILE_SIZE + threadIdx.y < N)
            tile_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tile_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

